#include "hip/hip_runtime.h"
/*
 * @file  cuCorrFrequency.cu
 * @brief A class performs cross correlation in frequency domain
 */

#include "cuCorrFrequency.h"
#include "cuAmpcorUtil.h"

/*
 * cuFreqCorrelator Constructor
 * @param imageNX height of each image
 * @param imageNY width of each image
 * @param nImages number of images in the batch
 * @param stream CUDA stream
 */
cuFreqCorrelator::cuFreqCorrelator(int imageNX, int imageNY, int nImages, hipStream_t stream_)
{

    int imageSize = imageNX*imageNY;
    int n[NRANK] ={imageNX, imageNY};

    // set up fft plans
    cufft_Error(hipfftPlanMany(&fftPlan, NRANK, n,
                              NULL, 1, imageSize,
                              NULL, 1, imageSize,
                              HIPFFT_C2C, nImages));

    stream = stream_;
    hipfftSetStream(fftPlan, stream);

    // set up work arrays
    workFM = new cuArrays<float2>(imageNX, imageNY, nImages);
    workFM->allocate();
    workFS = new cuArrays<float2>(imageNX, imageNY, nImages);
    workFS->allocate();
    workT = new cuArrays<float2> (imageNX, imageNY, nImages);
    workT->allocate();
}

/// destructor
cuFreqCorrelator::~cuFreqCorrelator()
{
    cufft_Error(hipfftDestroy(fftPlan));
    workFM->deallocate();
    workFS->deallocate();
    workT->deallocate();
}


/**
 * Execute the cross correlation
 * @param[in] templates the reference windows
 * @param[in] images the search windows
 * @param[out] results the correlation surfaces
 */

void cuFreqCorrelator::execute(cuArrays<float2> *templates, cuArrays<float2> *images, cuArrays<float> *results)
{
    // pad the reference windows to the the size of search windows
    // note: cuArraysCopyPadded has both float and float2 implementations
    cuArraysCopyPadded(templates, workT, stream);
    // forward fft to frequency domain
    cufft_Error(hipfftExecC2C(fftPlan, workT->devData, workFM->devData, HIPFFT_FORWARD));
    cufft_Error(hipfftExecC2C(fftPlan, images->devData, workFS->devData, HIPFFT_FORWARD));
    // cufft doesn't normalize, so manually get the image size for normalization
    float coef = 1.0/(images->size);
    // multiply reference with secondary windows in frequency domain
    cuArraysElementMultiplyConjugate(workFM, workFS, coef, stream);
    // backward fft to get correlation surface in time domain
    cufft_Error(hipfftExecC2C(fftPlan, workFM->devData, workT->devData, HIPFFT_BACKWARD));
    // extract to get proper size of correlation surface
    cuArraysCopyExtractC2A(workT, results, make_int2(0, 0), stream);
    // all done
}

// a = a^* * b
inline __device__ float2 cuMulConj(float2 a, float2 b)
{
    return make_float2(a.x*b.x + a.y*b.y, -a.y*b.x + a.x*b.y);
}

// cuda kernel for cuArraysElementMultiplyConjugate
__global__ void cudaKernel_elementMulConjugate(float2 *ainout, float2 *bin, int size, float coef)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < size) {
        hipComplex prod;
        prod = cuMulConj(ainout[idx], bin[idx]);
        ainout [idx] = prod*coef;
    }
}

/**
 * Perform multiplication of coef*Conjugate[image1]*image2 for each element
 * @param[inout] image1, the first image
 * @param[in] image2, the secondary image
 * @param[in] coef, usually the normalization factor
 */
void cuArraysElementMultiplyConjugate(cuArrays<float2> *image1, cuArrays<float2> *image2, float coef, hipStream_t stream)
{
    int size = image1->getSize();
    int threadsperblock = NTHREADS;
    int blockspergrid = IDIVUP (size, threadsperblock);
    cudaKernel_elementMulConjugate<<<blockspergrid, threadsperblock, 0, stream>>>(image1->devData, image2->devData, size, coef );
    getLastCudaError("cuArraysElementMultiply error\n");
}
//end of file
